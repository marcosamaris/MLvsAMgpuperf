
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void matMul(float* Pd, float* Md, float* Nd, int Width, int Tile_Width) {
  float Pvalue = 0.0;

  int j = blockIdx.x * Tile_Width + threadIdx.x;
  int i = blockIdx.y * Tile_Width + threadIdx.y;

  for (int k = 0; k < Width; ++k) {
    Pvalue += Md[j * Width + k] * Nd[k * Width + i];
  }

  Pd[j * Width + i] = Pvalue;
}


// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int k = 0; k < size; ++k) {
     data[k] = (float)drand48();
  }
}

int main(int argc, char* argv[])
{

  if (argc != 5) {
    fprintf(stderr, "Syntax: %s <matrix size> < Block_size> <CacheConfL1>  <device> \n", argv[0]);
    return EXIT_FAILURE;
  }


  int Width = atoi(argv[1]);
  int BlockSize = atoi(argv[2]);
  int CacheConfL1 = atoi(argv[3]);
  int devId = atoi(argv[4]);

  checkCuda( hipSetDevice(devId) );
  hipDeviceReset();

  // allocate host memory for matrices M and N
  printf("Allocate host memory for matrices M and N...\n");
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));

  // set seed for drand48()
  srand48(42);

  // initialize host matrices
  printf("Initialize host matrices...\n");
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  // allocate device matrices (linearized)
  printf("Allocate device matrices (linearized)...\n");
  float* Md = NULL; 
  float* Nd = NULL;
  float* Pd = NULL;
  checkCuda( hipMalloc((void**) &Md, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Nd, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Pd, Width * Width * sizeof(float)) );

  // copy host memory to device
  checkCuda( hipMemcpy(Md, M, Width*Width*sizeof(float), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(Nd, N, Width*Width*sizeof(float), hipMemcpyHostToDevice) );

  // execute the kernel
  printf("Execute the kernel...\n");

  if (CacheConfL1 == 1){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferShared);
  }
  else if (CacheConfL1 == 2){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferEqual);
  }
  else if (CacheConfL1 == 3){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferL1);
  }
  else {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferNone);
  }

  int GridSize = (Width + BlockSize-1) / BlockSize;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(BlockSize, BlockSize);

  hipProfilerStart(); 
  matMul<<< gridDim, blockDim >>>(Pd, Md, Nd, Width, BlockSize);
  hipProfilerStop();

  // copy result from device to host
  checkCuda( hipMemcpy( P, Pd, Width * Width * sizeof(float),hipMemcpyDeviceToHost) );

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId) );
  printf("Device: %s\n", prop.name);

  // clean up memory
  free(M);
  free(N);
  free(P);

  checkCuda( hipFree(Md) );
  checkCuda( hipFree(Nd) );
  checkCuda( hipFree(Pd) );

  return 0;

}

